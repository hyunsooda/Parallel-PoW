#include "hip/hip_runtime.h"
#include <string.h>
#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""

#include "sha256.h"

using namespace std;

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do {\
		(x); \
		hipError_t e = hipGetLastError(); \
		if (hipSuccess != e) { \
			printf("cuda failure \"%s\" at %s:%d\n", \
			       hipGetErrorString(e), \
			       __FILE__, __LINE__); \
			exit(1); \
		} \
	} while (0)
#endif


const int targetBit = 6;
const int operationPerThread = 900; // you have to adjust it whenever execute for purpose(hash difficulty) what you want

__device__  int my_strlen(char *string) {
	int cnt = 0;
	while (string[cnt] != '\0') {
		++cnt;
	}
	return cnt;
}


__device__ int _atoi(char const *c) {

	int value = 0;
	int positive = 1;

	if (*c == '\0')
		return 0;

	if (*c == '-')
		positive = -1;

	while (*c) {
		if (*c > '0' && *c < '9')
			value = value * 10 + *c - '0';
		c++;
	}

	return value*positive;
}

__device__  void reverseString(char* s) {
	int size = my_strlen(s);
	char temp;

	for (int i = 0; i < size / 2; i++) {
		temp = s[i];
		s[i] = s[(size - 1) - i];
		s[(size - 1) - i] = temp;
	}
}

__device__ char* _itoa(long long val, char * buf, int radix) {

	char* p = buf;

	while (val) {

		if (radix <= 10)
			*p++ = (val % radix) + '0';

		else {
			int t = val % radix;
			if (t <= 9)
				*p++ = t + '0';
			else
				*p++ = t - 10 + 'a';
		}

		val /= radix;
	}

	*p = '\0';
	reverseString(buf);
	//reverse(buf); 
	return buf;
}

__device__  void my_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[++i] != '\0');
}

__device__  void my_strcat(char *c, char *m) {
	while (*c != '\0') { c++; }
	while (*m != '\0') { *c++ = *m++; }
	*c = '\0';
}


__host__ void genRandomTransactionHash(char *dest, int length) {
	srand(time(NULL));

	char charset[] = "0123456789"
		"abcdefghijklmnopqrstuvwxyz"
		"ABCDEFGHIJKLMNOPQRSTUVWXYZ";

	while (length-- > 1) {
		int index = (double)rand() / RAND_MAX * (sizeof charset - 1);
		*dest++ = charset[index];
	}
	*dest = '\0';
}

__device__  void prepareData(char* timestamp, char* transactionHash, char* nonce, char* output) {
	char input[200] = { 0, };
	char out[100];
	// sprintf(output, "%lu%s%s", timestamp, transactionHash, nonce);
	
	my_strcat(input, timestamp);
	my_strcat(input, transactionHash);
	my_strcat(input, nonce);
	sha256(input, my_strlen(input), output);
	
}

__device__ int count(char *c)
{
	int i, count;
	count = 0;
	for (i = 0; c[i] != NULL; i++)
	{
		if (c[i] == '0')
			count++;
		else
			break;
	}
	return count;
}

__device__ long long getNonce(int bx, int tx) {
	int txn, bxn;
	long long n;

	n = 0;
	bxn = operationPerThread;

	if (bx) {
		bxn = bxn * blockDim.x; // blockDim.x
		bxn *= bx;
	}

	txn = operationPerThread * tx;
	n += bxn;
	n += txn;

	return n;
}

__global__ void mine(char* timestamp, char* transaction, char* ret_nonce, char* ret_hash) {
	unsigned char isSuccess;
	char output[100];
	char nonce_str[100];
	long long numOfTrial = 0;
	long long nonce;

	nonce = getNonce((int)blockIdx.x, (int)threadIdx.x);

	for (int k = 0; k < operationPerThread; k++) {
		_itoa(nonce, nonce_str, 10);	
		prepareData(timestamp, transaction, nonce_str, output);	
		if (count(output) >= targetBit) {
			_itoa(numOfTrial, nonce_str, 10);
			my_strcpy(ret_nonce, nonce_str);
			my_strcpy(ret_hash, output);
			return;
		}	
		
		nonce--;
		numOfTrial++;
	}
}


__global__ void foo(char* ret_nonce) {
	char* input = "asdasd";
	char output[100];
	for (int i = 0; i < operationPerThread; i++) {
		prepareData("123123123", "123123123123", "123123123", output);
	}
}


__host__ char* myitoa(long long val, char * buf, int radix) {

	char* p = buf;

	while (val) {

		if (radix <= 10)
			*p++ = (val % radix) + '0';

		else {
			int t = val % radix;
			if (t <= 9)
				*p++ = t + '0';
			else
				*p++ = t - 10 + 'a';
		}

		val /= radix;
	}

	*p = '\0';

	//reverse(buf); 
	return buf;
}
__host__  void strcc(char *c, char *m) {
	while (*c != '\0') { c++; }
	while (*m != '\0') { *c++ = *m++; }
	*c = '\0';
}

__host__  int mystrlen(char *string) {
	int cnt = 0;
	while (string[cnt] != '\0') {
		++cnt;
	}
	return cnt;
}

int main(int argc, char* argv[]) {
	const int numOfBlocks = 10;  // you have to adjust it whenever execute for purpose(hash difficulty) what you want
	const int numOfThreads = 1024;
	char transaction[50];
	char* d_nonce;
	char* h_nonce;
	char* d_hash;
	char* h_hash;
	char* d_transaction;
	char* d_timestamp;
	dim3 blocks(numOfBlocks, 1, 1);
	dim3 threads(numOfThreads, 1, 1);
	unsigned long t = (unsigned long)time(NULL);
	char timestamp[20];
	hipEvent_t start, stop;
	float ms = 0;
	CUDA_CHECK(hipEventCreate(&start));
	CUDA_CHECK(hipEventCreate(&stop));

	sprintf(timestamp, "%lu", t);
	genRandomTransactionHash(transaction, sizeof(transaction));

	h_nonce = (char*)malloc(sizeof(char) * 100);
	h_hash = (char*)malloc(sizeof(char) * 100);
	memset(h_nonce, 0, sizeof(char) * 100);
	memset(h_hash, 0, sizeof(char) * 100);
	CUDA_CHECK(hipEventRecord(start));
	CUDA_CHECK(hipMalloc((void**)&d_nonce, sizeof(char) * 100));
	CUDA_CHECK(hipMalloc((void**)&d_hash, sizeof(char) * 100));
	CUDA_CHECK(hipMalloc((void**)&d_transaction, sizeof(transaction)));
	CUDA_CHECK(hipMalloc((void**)&d_timestamp, sizeof(timestamp)));
	CUDA_CHECK(hipMemcpy(d_nonce, h_nonce, sizeof(char) * 100, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_hash, h_hash, sizeof(char) * 100, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_transaction, transaction, sizeof(transaction), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_timestamp, timestamp, sizeof(timestamp), hipMemcpyHostToDevice));
	
	
	mine<<<blocks, threads >>> (d_timestamp, d_transaction, d_nonce, d_hash);
	CUDA_CHECK(hipPeekAtLastError());
	CUDA_CHECK(hipMemcpy(h_nonce, d_nonce, sizeof(char) * 100, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_hash, d_hash, sizeof(char) * 100, hipMemcpyDeviceToHost));
	printf("hash : %s\n", h_hash);
	CUDA_CHECK(hipEventRecord(stop));
	hipEventSynchronize(stop);
	CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
	printf("The time duration : %fms\n", ms);

}


